#include "hip/hip_runtime.h"

#include <cstdio>
#ifndef HPCG_NO_MPI
#include <mpi.h>
#include "mytimer.hpp"
#endif

#include <cassert>
#include <hip/hip_runtime_api.h>
#include "ComputeDotProduct_cuda.hpp"

__global__ void dotProductKernel(const double* x, const double* y, double* partialResults, local_int_t n) {
    extern __shared__ double sdata[];

    // Calculate thread index
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared memory
    sdata[tid] = 0.0;

    // Perform partial dot product
    if (i < n) {
        sdata[tid] = x[i] * y[i];
    }
    __syncthreads();

    // Parallel reduction within the block
    // blockDim is 256
    // one block's(256) result is reduced to sdata[0]
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the result for this block to the partial results array
    if (tid == 0) {
        partialResults[blockIdx.x] = sdata[0];
        // printf("blockIdx %d, partialResults[blockIdx.x] %f\n", blockIdx.x, partialResults[blockIdx.x]);
    }
}

int ComputeDotProduct_cuda(const local_int_t n, const Vector & x, const Vector & y,
    double & result, double & time_allreduce) {
    assert(x.localLength >= n); // Test vector lengths
    assert(y.localLength >= n);

    // Allocate device memory
    double* d_x;
    double* d_y;
    double* d_partialResults;
    hipMalloc((void**)&d_x, n * sizeof(double));
    hipMalloc((void**)&d_y, n * sizeof(double));

    // Copy vectors to device
    hipMemcpy(d_x, x.values, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y.values, n * sizeof(double), hipMemcpyHostToDevice);

    // Calculate grid and block sizes
    int blockSize = 256; // You can adjust this value
    // int blockSize = 1024; // You can adjust this value
    // int blockSize = 2048; // You can adjust this value
    int gridSize = (n + blockSize - 1) / blockSize;
    hipMalloc((void**)&d_partialResults, gridSize * sizeof(double));

    // Launch kernel
    dotProductKernel<<<gridSize, blockSize, blockSize * sizeof(double)>>>(d_x, d_y, d_partialResults, n);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    // Copy partial results back to host
    double* h_partialResults = new double[gridSize];
    hipMemcpy(h_partialResults, d_partialResults, gridSize * sizeof(double), hipMemcpyDeviceToHost);

    // Final reduction on host
    double local_result = 0.0;
    for (int i = 0; i < gridSize; ++i) {
        local_result += h_partialResults[i];
    }

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_partialResults);
    delete[] h_partialResults;

#ifndef HPCG_NO_MPI
    // Use MPI's reduce function to collect all partial sums
    double t0 = mytimer();
    double global_result = 0.0;
    MPI_Allreduce(&local_result, &global_result, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
    result = global_result;
    time_allreduce += mytimer() - t0;
#else
    time_allreduce += 0.0;
    result = local_result;
#endif

    return 0;
}

// __global__ void dotProductKernel(const double* x, const double* y, double* partialResults, local_int_t n) {
//     extern __shared__ double sdata[];
// 
//     // Calculate thread index
//     unsigned int tid = threadIdx.x;
//     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
// 
//     // Initialize shared memory
//     sdata[tid] = 0.0;
// 
//     // Perform partial dot product
//     if (i < n) {
//         sdata[tid] = x[i] * y[i];
//     }
//     __syncthreads();
// 
//     // Parallel reduction within the block
//     // blockDim is 256
//     // one block's(256) result is reduced to sdata[0]
//     for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
//         if (tid < s) {
//             sdata[tid] += sdata[tid + s];
//         }
//         __syncthreads();
//     }
// 
//     // Write the result for this block to the partial results array
//     if (tid == 0) {
//         partialResults[blockIdx.x] = sdata[0];
//         // print the address of sdata[0] and blockIdx.x
//         printf("sdata[0] address: %p ", &sdata[0]);
//         // printf("blockIdx.x: %d\n", blockIdx.x);
//         // printf("blockIdx %d, partialResults[blockIdx.x] %f\n", blockIdx.x, partialResults[blockIdx.x]);
//     }
// }
// 
// int ComputeDotProduct_cuda(const local_int_t n, const Vector & x, const Vector & y,
//     double & result, double & time_allreduce) {
//     assert(x.localLength >= n); // Test vector lengths
//     assert(y.localLength >= n);
// 
//     // Allocate device memory
//     double* d_x;
//     double* d_y;
//     double* d_partialResults;
//     hipMalloc((void**)&d_x, n * sizeof(double));
//     hipMalloc((void**)&d_y, n * sizeof(double));
// 
//     // Copy vectors to device
//     hipMemcpy(d_x, x.values, n * sizeof(double), hipMemcpyHostToDevice);
//     hipMemcpy(d_y, y.values, n * sizeof(double), hipMemcpyHostToDevice);
// 
//     // Calculate grid and block sizes
//     int blockSize = 256; // You can adjust this value
//     // int blockSize = 1024; // You can adjust this value
//     // int blockSize = 2048; // You can adjust this value
//     int gridSize = (n + blockSize - 1) / blockSize;
//     hipMalloc((void**)&d_partialResults, gridSize * sizeof(double));
// 
//     // Launch kernel
//     dotProductKernel<<<gridSize, blockSize, blockSize * sizeof(double)>>>(d_x, d_y, d_partialResults, n);
// 
//     hipError_t err = hipGetLastError();
//     if (err != hipSuccess) {
//         printf("CUDA Error: %s\n", hipGetErrorString(err));
//     }
//     // Copy partial results back to host
//     double* h_partialResults = new double[gridSize];
//     hipMemcpy(h_partialResults, d_partialResults, gridSize * sizeof(double), hipMemcpyDeviceToHost);
// 
//     // Final reduction on host
//     double local_result = 0.0;
//     for (int i = 0; i < gridSize; ++i) {
//         local_result += h_partialResults[i];
//     }
// 
//     // Free device memory
//     hipFree(d_x);
//     hipFree(d_y);
//     hipFree(d_partialResults);
//     delete[] h_partialResults;
// 
// #ifndef HPCG_NO_MPI
//     // Use MPI's reduce function to collect all partial sums
//     double t0 = mytimer();
//     double global_result = 0.0;
//     MPI_Allreduce(&local_result, &global_result, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
//     result = global_result;
//     time_allreduce += mytimer() - t0;
// #else
//     time_allreduce += 0.0;
//     result = local_result;
// #endif
// 
//     return 0;
// }


